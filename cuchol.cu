#include "hip/hip_runtime.h"
#include "device_launch_paraMeters.h"

#include<iostream>
#include <fstream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "Utilities.cuh"

#define prec_save 10

/******************************************/
/* SET HERMITIAN POSITIVE DEFINITE MATRIX */
/******************************************/
// --- Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void setPDMatrix(double * __restrict h_A, const int N) {

    // --- Initialize random seed
    srand(time(NULL));

    double *h_A_temp = (double *)malloc(N * N * sizeof(double));

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            h_A_temp[i * N + j] = (float)rand() / (float)RAND_MAX;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) 
            h_A[i * N + j] = 0.5 * (h_A_temp[i * N + j] + h_A_temp[j * N + i]);

    for (int i = 0; i < N; i++) h_A[i * N + i] = h_A[i * N + i] + N;

}

/************************************/
/* SAVE REAL ARRAY FROM CPU TO FILE */
/************************************/
template <class T>
void saveCPUrealtxt(const T * h_in, const char *filename, const int M) {

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}

/************************************/
/* SAVE REAL ARRAY FROM GPU TO FILE */
/************************************/
template <class T>
void saveGPUrealtxt(const T * d_in, const char *filename, const int M) {

    T *h_in = (T *)malloc(M * sizeof(T));

    gpuErrchk(hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost));

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}

/********/
/* MAIN */
/********/
int main(){

    const int N = 1000;

    // --- CUDA solver initialization
    hipsolverHandle_t solver_handle;
    cusolveSafeCall(hipsolverDnCreate(&solver_handle));

    // --- CUBLAS initialization
    hipblasHandle_t cublas_handle;
    cublasSafeCall(hipblasCreate(&cublas_handle));

    /***********************/
    /* SETTING THE PROBLEM */
    /***********************/
    // --- Setting the host, N x N matrix
    double *h_A = (double *)malloc(N * N * sizeof(double));
    setPDMatrix(h_A, N);

    // --- Allocate device space for the input matrix 
    double *d_A; gpuErrchk(hipMalloc(&d_A, N * N * sizeof(double)));

    // --- Move the relevant matrix from host to device
    gpuErrchk(hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice));

    /****************************************/
    /* COMPUTING THE CHOLESKY DECOMPOSITION */
    /****************************************/
    // --- cuSOLVE input/output parameters/arrays
    int work_size = 0;
    int *devInfo;           gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

    // --- CUDA CHOLESKY initialization
    cusolveSafeCall(hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, &work_size));

    // --- CUDA POTRF execution
    double *work;   gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));
    cusolveSafeCall(hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, work, work_size, devInfo));
    int devInfo_h = 0;  gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) std::cout << "Unsuccessful potrf execution\n\n" << "devInfo = " << devInfo_h << "\n\n";

    // --- At this point, the lower triangular part of A contains the elements of L. 
    /***************************************/
    /* CHECKING THE CHOLESKY DECOMPOSITION */
    /***************************************/
    saveCPUrealtxt(h_A, "D:\\Project\\solveSquareLinearSystemCholeskyCUDA\\solveSquareLinearSystemCholeskyCUDA\\h_A.txt", N * N);
    saveGPUrealtxt(d_A, "D:\\Project\\solveSquareLinearSystemCholeskyCUDA\\solveSquareLinearSystemCholeskyCUDA\\d_A.txt", N * N);

    cusolveSafeCall(hipsolverDnDestroy(solver_handle));

    return 0;

}